#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include <vector>
#include "utils.hpp"
#include <hip/hip_fp16.h>
#include "convert_cuda_impl.hpp"
#define THREADS_PER_BLOCK 256
#define ELEMENTS_PER_THREAD 4


namespace CudaImpl
{

    __global__ void fp32ToFp16Kernel(const float* input_data, const int input_size, half* output_data)
    {
        int start = ELEMENTS_PER_THREAD * THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
        float value[ELEMENTS_PER_THREAD];
      
        int id = start;
        #pragma unroll
        for (int i = 0; i < ELEMENTS_PER_THREAD; i++)
        {
            if (id < input_size)
            {
                value[i] = input_data[id];
                id += THREADS_PER_BLOCK;
            }
        }
      
        id = start;
        #pragma unroll
        for (int i = 0; i < ELEMENTS_PER_THREAD; i++)
        {
            if (id < input_size)
            {
                output_data[id] = half(value[i]);
                id += THREADS_PER_BLOCK;
            }
        }        
    }

    __global__ void fp16ToFp32Kernel(const half* input_data, const int input_size, float* output_data)
    {
        int start = ELEMENTS_PER_THREAD * THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
        half value[ELEMENTS_PER_THREAD];
      
        int id = start;
        #pragma unroll
        for (int i = 0; i < ELEMENTS_PER_THREAD; i++)
        {
            if (id < input_size)
            {
                value[i] = input_data[id];
                id += THREADS_PER_BLOCK;
            }
        }
      
        id = start;
        #pragma unroll
        for (int i = 0; i < ELEMENTS_PER_THREAD; i++)
        {
            if (id < input_size)
            {
                output_data[id] = float(value[i]);
                id += THREADS_PER_BLOCK;
            }
        }        
    }

    void ConvertFp16ToFp32CudaImpl(const void* input, const int inputSize, void* output, hipStream_t stream)
    {
        int blockSize = THREADS_PER_BLOCK;
        int gridSize = inputSize / (blockSize * ELEMENTS_PER_THREAD) + 1;
        fp16ToFp32Kernel<<<gridSize, blockSize, 0, stream>>>((half*)input, inputSize, (float*)output);
        hipError_t cudastatus = hipGetLastError();
        CHECK_ASSERT(cudastatus == hipSuccess, "launch failed: %s\n", hipGetErrorString(cudastatus));
    }

    void ConvertFp32ToFp16CudaImpl(const void* input, const int inputSize, void* output, hipStream_t stream)
    {
        int blockSize = THREADS_PER_BLOCK;
        int gridSize = inputSize / (blockSize * ELEMENTS_PER_THREAD) + 1;
        fp32ToFp16Kernel<<<gridSize, blockSize, 0, stream>>>((float*)input, inputSize, (half*)output);
        hipError_t cudastatus = hipGetLastError();
        CHECK_ASSERT(cudastatus == hipSuccess, "launch failed: %s\n", hipGetErrorString(cudastatus));
    }    
}

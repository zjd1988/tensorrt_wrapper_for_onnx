#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include <vector>
#include "utils.hpp"
#include <hipcub/hipcub.hpp>
#include "nonzero_cuda_impl.hpp"
#define THREADS_PER_BLOCK 256


namespace CudaImpl
{

    hipError_t NonZeroCalcPrefixSumTempStorageBytes(int* prefix_counts, int number_of_blocks,
            size_t& temp_storage_bytes, hipStream_t stream)
    {
        temp_storage_bytes = 0;
        return hipcub::DeviceScan::InclusiveSum(
            nullptr, temp_storage_bytes, prefix_counts, prefix_counts, number_of_blocks, stream);
    }

    hipError_t NonZeroInclusivePrefixSum(void* d_temp_storage, size_t temp_storage_bytes, int* prefix_counts,
            int number_of_blocks, hipStream_t stream)
    {
        return hipcub::DeviceScan::InclusiveSum(
            d_temp_storage, temp_storage_bytes, prefix_counts, prefix_counts, number_of_blocks, stream);
    }

    __global__ void NonZeroCountEachBlockKernel(const unsigned char* x, int x_size, int* count_in_blocks)
    {
        typedef hipcub::BlockReduce<int, THREADS_PER_BLOCK, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduceT;
        __shared__ typename BlockReduceT::TempStorage temp_storage;

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int nz = 0;
        if (index < x_size && x[index] == 1)
            ++nz;

        int count = BlockReduceT(temp_storage).Sum(nz);

        if (threadIdx.x == 0) {
            count_in_blocks[blockIdx.x] = count;
        }
    }

    __global__ void NonZeroOutputPositionsKernel(const unsigned char* x, const int x_size, const int* prefix_counts, int* results)
    {
        typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScanT;
        __shared__ typename BlockScanT::TempStorage temp_storage;

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int nz = 0;
        if (index < x_size && x[index] == 1)
            ++nz;
        int pos_in_block = 0;
        BlockScanT(temp_storage).InclusiveSum(nz, pos_in_block);

        int result_position = ((blockIdx.x == 0) ? 0 : prefix_counts[blockIdx.x - 1]) + pos_in_block - nz;

        if (index < x_size && x[index] == 1) {
            results[result_position] = index;
            // printf("result_position %d index %d\n");
        }
    }

    void NoneZeroCudaImpl(const unsigned char* input, const int inputSize, int* output, hipStream_t stream)
    {
        const int threadCount = THREADS_PER_BLOCK;
        int blockSize = threadCount;
        int gridSize = (inputSize + blockSize - 1) / blockSize;
        int index = inputSize / 2;
        size_t tempStorageBytes = 0;
        NonZeroCountEachBlockKernel<<<gridSize, blockSize, 0, stream>>>(input, inputSize, output + index);
        NonZeroCalcPrefixSumTempStorageBytes(output + index, gridSize, tempStorageBytes, stream);
        NonZeroInclusivePrefixSum(output + index + gridSize, tempStorageBytes, output + index, gridSize, stream);
        NonZeroOutputPositionsKernel<<<gridSize, blockSize, 0, stream>>>(input, inputSize, output + index, output);
        hipError_t cudastatus = hipGetLastError();
        CHECK_ASSERT(cudastatus == hipSuccess, "launch failed: %s\n", hipGetErrorString(cudastatus));
    }
}

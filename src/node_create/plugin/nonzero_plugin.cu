#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include "utils.hpp"
#include "nonzero_plugin.hpp"
#include <hipcub/hipcub.hpp>

#define THREADS_PER_BLOCK 256

NonZeroPlugin::NonZeroPlugin()
{
}

NonZeroPlugin::~NonZeroPlugin()
{
}

// create the plugin at runtime from a byte stream
NonZeroPlugin::NonZeroPlugin(const void* data, size_t length)
{
    assert(length == sizeof(inputSize));
    inputSize = *reinterpret_cast<const int*>(data);
}


/********************************************************************************
/////////////////inherited from nvinfer1::IPluginV2Ext///////////////////////////
********************************************************************************/
// Return the DataType of the plugin output at the requested index
nvinfer1::DataType NonZeroPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
{
    return nvinfer1::DataType::kINT32;
}

// Return true if output tensor is broadcast across a batch.
bool NonZeroPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
{
    return false;
}

// Return true if plugin can use input that is broadcast across batch without replication.
bool NonZeroPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
{
    return false;
}

void NonZeroPlugin::configurePlugin(const nvinfer1::PluginTensorDesc* in, int nbInput, const nvinfer1::PluginTensorDesc* out, int nbOutput)
{
    // int pos = 0;
    // in[pos]->format = nvinfer1::DataType::kINT8;
    // return;
}

// Attach the plugin object to an execution context and grant the plugin the access to some context resource.
void NonZeroPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, nvinfer1::IGpuAllocator* gpuAllocator)
{
}

// Detach the plugin object from its execution context.
void NonZeroPlugin::detachFromContext()
{
    return;
}

// Clone the plugin
nvinfer1::IPluginV2IOExt* NonZeroPlugin::clone() const
{
    NonZeroPlugin *p = new NonZeroPlugin();
    p->inputSize = inputSize;
    p->setPluginNamespace(mPluginNamespace);
    return p;
}
/********************************************************************************
/////////////////inherited from nvinfer1::IPluginV2//////////////////////////////
********************************************************************************/

const char* NonZeroPlugin::getPluginType() const
{
    return "NonZero_TRT";
}

const char* NonZeroPlugin::getPluginVersion() const
{
    return "1";
}

int NonZeroPlugin::getNbOutputs() const
{
    return 1;
}

nvinfer1::Dims NonZeroPlugin::getOutputDimensions(int index, const nvinfer1::Dims* inputs, int nbInputDims)
{
    CHECK_ASSERT(nbInputDims == 1 && index == 0, "nonzero only support 1 input!\n");
    inputSize = 1;
    nvinfer1::Dims dims;
    for(int i = 0; i < inputs[0].nbDims; i++)
    {
        inputSize *= inputs[0].d[i];
        dims.d[i] = inputs[0].d[i];
    }
    dims.nbDims = inputs[0].nbDims;
    // Output dimensions
    return dims;
}

int NonZeroPlugin::initialize()
{ 
    return 0;
}
void NonZeroPlugin::terminate()
{
    return;
}

size_t NonZeroPlugin::getWorkspaceSize(int maxBatchSize) const
{
    return 0;
}

hipError_t NonZeroCalcPrefixSumTempStorageBytes(int* prefix_counts, int number_of_blocks,
            size_t& temp_storage_bytes, hipStream_t stream)
{
  temp_storage_bytes = 0;
  return hipcub::DeviceScan::InclusiveSum(
      nullptr, temp_storage_bytes, prefix_counts, prefix_counts, number_of_blocks, stream);
}

hipError_t NonZeroInclusivePrefixSum(void* d_temp_storage, size_t temp_storage_bytes, int* prefix_counts,
             int number_of_blocks, hipStream_t stream)
{
  return hipcub::DeviceScan::InclusiveSum(
      d_temp_storage, temp_storage_bytes, prefix_counts, prefix_counts, number_of_blocks, stream);
}

__global__ void NonZeroCountEachBlockKernel(const unsigned char* x, int x_size, int* count_in_blocks) {
    typedef hipcub::BlockReduce<int, THREADS_PER_BLOCK, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduceT;
    __shared__ typename BlockReduceT::TempStorage temp_storage;
  
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int nz = 0;
    if (index < x_size && x[index] == 1)
        ++nz;

    int count = BlockReduceT(temp_storage).Sum(nz);
  
    if (threadIdx.x == 0) {
      count_in_blocks[blockIdx.x] = count;
    }
}

__global__ void NonZeroOutputPositionsKernel(const unsigned char* x, const int x_size, const int* prefix_counts, int* results)
{
    typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScanT;
    __shared__ typename BlockScanT::TempStorage temp_storage;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int nz = 0;
    if (index < x_size && x[index] == 1)
        ++nz;
    int pos_in_block = 0;
    BlockScanT(temp_storage).InclusiveSum(nz, pos_in_block);

    int result_position = ((blockIdx.x == 0) ? 0 : prefix_counts[blockIdx.x - 1]) + pos_in_block - nz;

    if (index < x_size && x[index] == 1) {
        results[result_position] = index;
        // printf("result_position %d index %d\n");
    }
}

void NonZeroPlugin::forwardGpu(const unsigned char *const * inputs, int* output, hipStream_t stream, int batchSize) {
    const int threadCount = 128;
    int blockSize = threadCount;
    int gridSize = (inputSize * batchSize + blockSize - 1) / blockSize;
    int index = inputSize / 2;
    size_t tempStorageBytes = 0;
    NonZeroCountEachBlockKernel<<<gridSize, blockSize, 0, stream>>>(inputs[0], inputSize * batchSize, output + index);
    NonZeroCalcPrefixSumTempStorageBytes(output + index, gridSize, tempStorageBytes, stream);
    NonZeroInclusivePrefixSum(output + index + gridSize, tempStorageBytes, output + index, gridSize, stream);
    NonZeroOutputPositionsKernel<<<gridSize, blockSize, 0, stream>>>(inputs[0], inputSize * batchSize, output + index, output);
}

int NonZeroPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
{
    //assert(batchSize == 1);
    //GPU
    forwardGpu((const unsigned char *const *)inputs, (int*)outputs[0], stream, batchSize);
    return 0;
}

size_t NonZeroPlugin::getSerializationSize() const
{  
    return sizeof(inputSize);
}

void NonZeroPlugin::serialize(void* buffer) const
{
    *reinterpret_cast<int*>(buffer) = inputSize;
}

void NonZeroPlugin::destroy()
{
    delete this;
}

// Set plugin namespace
void NonZeroPlugin::setPluginNamespace(const char* pluginNamespace)
{
    mPluginNamespace = pluginNamespace;
}

const char* NonZeroPlugin::getPluginNamespace() const
{
    return mPluginNamespace;
}



/********************************************************************************
/////////////////inherited from nvinfer1::IPluginCreator/////////////////////////
********************************************************************************/
NonZeroPluginCreator::NonZeroPluginCreator()
{
    mPluginAttributes.clear();
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* NonZeroPluginCreator::getPluginName() const
{
    return "NonZero_TRT";
}

const char* NonZeroPluginCreator::getPluginVersion() const
{
    return "1";
}

const nvinfer1::PluginFieldCollection* NonZeroPluginCreator::getFieldNames()
{
    return &mFC;
}

nvinfer1::IPluginV2IOExt* NonZeroPluginCreator::createPlugin(const char* name, const nvinfer1::PluginFieldCollection* fc)
{
    NonZeroPlugin* obj = new NonZeroPlugin();
    obj->setPluginNamespace(name);
    return obj;
}

nvinfer1::IPluginV2IOExt* NonZeroPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
{
    // This object will be deleted when the network is destroyed, which will
    // call NonZeroPlugin::destroy()
    NonZeroPlugin* obj = new NonZeroPlugin(serialData, serialLength);
    obj->setPluginNamespace(name);
    return obj;
}


REGISTER_TENSORRT_PLUGIN(NonZeroPluginCreator);